#define N 4000

#define DIV_UP(a, b) ( ((a) + (b) - 1) / (b) )

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void matrixMult (float *a, float *b, float *c, int width) {
 int k = 0;
 float sum = 0.0;
 int col = threadIdx.x + blockDim.x * blockIdx.x;
 int row = threadIdx.y + blockDim.y * blockIdx.y;
 if(col < width && row < width) {
   for (k = 0; k < width; k++)
      sum += a[row * width + k] * b[k * width + col];
   c[row * width + col] = sum;
 }
}
int main() {
 //float a[N][N], b[N][N], c[N][N];
 float *dev_a, *dev_b, *dev_c;
float *a,*b,*c;
a = (float*) malloc(N*N*sizeof(float));
b = (float*) malloc(N*N*sizeof(float));
c = (float*) malloc(N*N*sizeof(float));

 // initialize matrices a and b with appropriate values
 for (int i = 0; i< N ; i++)
 for (int j=0 ; j<N ; j++)
 {
   a[i*N+j] = 1;
   b[i*N+j] = 1;
 }
 int size = N * N * sizeof(float);
 hipMalloc((void **) &dev_a, size);
 hipMalloc((void **) &dev_b, size);
 hipMalloc((void **) &dev_c, size);
 hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
 int NumThreads = 32;
 dim3 dimGrid(DIV_UP(N,NumThreads), DIV_UP(N,NumThreads));
 dim3 dimBlock(NumThreads, NumThreads);

 matrixMult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, N);
//measure performance
 hipError_t error;
	hipDeviceSynchronize();
    hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));

    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));



 int nIter = 10;

    for (int j = 0; j < nIter; j++)
    {

		matrixMult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, N);

	}
	
	
    // Record the stop event
    error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));


    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    	printf ("msec %f\n",msecPerMatrixMul); 
    	
   //end measure performance
 hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
/*
 for (int i = 0; i< N ; i++){
 for (int j=0 ; j<N ; j++)
 {
   printf("%f , ", c[i][j]);

 }
 printf ("\n");
 }
 */
printf("%f, %f \n",c[0],c[N*N-1] );

 hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
}
