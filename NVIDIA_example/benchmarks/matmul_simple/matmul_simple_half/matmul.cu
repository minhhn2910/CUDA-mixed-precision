#include "hip/hip_runtime.h"
#define N 4000
#define DIV_UP(a, b) ( ((a) + (b) - 1) / (b) )
#include <stdio.h>
#include <hip/hip_fp16.h>
#include "newhalf.hpp"
#include "half_operator_overload.cuh"
__global__ void matrixMult (__half *a, __half *b, __half *c, int width) {
 int k = 0;
 half sum = __float2half(0);
 int col = threadIdx.x + blockDim.x * blockIdx.x;
 int row = threadIdx.y + blockDim.y * blockIdx.y;
 if(col < width && row < width) {
   for (k = 0; k < width; k++)
      sum += a[row * width + k] * b[k * width + col];
   c[row * width + col] = sum;
 }
}
int main() {
 //half_float::half a[N][N], b[N][N], c[N][N];
 half *dev_a, *dev_b, *dev_c;
//float *dev_a, *dev_b, *dev_c;
half_float::half *a,*b,*c;
a = (half_float::half*) malloc(N*N*sizeof(half_float::half));
b = (half_float::half*) malloc(N*N*sizeof(half_float::half));
c = (half_float::half*) malloc(N*N*sizeof(half_float::half));

 // initialize matrices a and b with appropri
 // initialize matrices a and b with appropriate values
 for (int i = 0; i< N ; i++)
 for (int j=0 ; j<N ; j++)
 {
   a[i*N +j] = half_float::half(0.1);
   b[i*N +j] = half_float::half(0.1);
 }
 
 int size = N * N * sizeof(half);
 hipMalloc((void **) &dev_a, size);
 hipMalloc((void **) &dev_b, size);
 hipMalloc((void **) &dev_c, size);
 hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
 int NumThreads = 32;
 dim3 dimGrid(DIV_UP(N,NumThreads), DIV_UP(N,NumThreads));
 dim3 dimBlock(NumThreads, NumThreads);
 matrixMult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, N);
 
 
 //measure performance
 hipError_t error;
	hipDeviceSynchronize();
    hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));

    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));



 int nIter = 10;

    for (int j = 0; j < nIter; j++)
    {

		matrixMult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, N);

	}
	

    // Record the stop event
    error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));


    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    	printf ("msec %f\n",msecPerMatrixMul); 
    	
   //end measure performance
 
 
 hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
 hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
 /*
  * for (int i = 0; i< N ; i++){
 for (int j=0 ; j<N ; j++)
 {
   printf("%f , ", float(c[i][j]));

 }
 printf ("\n");
 }*/

}
