#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

//#include "../../../include/fast_math.cuh"
#define SLOW_MATH

#ifdef SLOW_MATH
  #include "../../../include/cuda_math.cuh"
#else //my approximate math lib
  #include "../../../include/fast_math.cuh"
#endif

#include <hip/hip_runtime_api.h>
#include <math.h>

#if defined(__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

// CUDA standard includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include "bodysystem.h"

#define SCATTER

#define SPEED 10
/*
typedef struct __device_builtin__ half2_3
{
    half2 x, y, z;
} half2_3 ;

typedef struct __device_builtin__ __builtin_align__(16) half2_4
{
    half2 x, y, z, w;
} half2_4;
*/
//typedef __device_builtin__ struct half2_3 half2_3;
//typedef __device_builtin__ struct half2_4 half2_4;
__constant__ float softeningSquared;
__constant__ double softeningSquared_fp64;

hipError_t setSofteningSquared(float softeningSq)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(softeningSquared),
                              &softeningSq,
                              sizeof(float), 0,
                              hipMemcpyHostToDevice);
}

hipError_t setSofteningSquared(double softeningSq)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(softeningSquared_fp64),
                              &softeningSq,
                              sizeof(double), 0,
                              hipMemcpyHostToDevice);
}

template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

template<typename T>
__device__ T rsqrt_T(T x)
{
    return rsqrt(x);
}

template<>
__device__ float rsqrt_T<float>(float x)
{
    return rsqrtf(x);
}

template<>
__device__ double rsqrt_T<double>(double x)
{
    return rsqrt(x);
}


// Macros to simplify shared memory addressing
#define SX(i) sharedPos[i+blockDim.x*threadIdx.y]
// This macro is only used when multithreadBodies is true (below)
#define SX_SUM(i,j) sharedPos[i+blockDim.x*j]

template <typename T>
__device__ T getSofteningSquared()
{
    return softeningSquared;
}
template <>
__device__ double getSofteningSquared<double>()
{
    return softeningSquared_fp64;
}

template <typename T>
struct DeviceData
{
    T *dPos[2]; // mapped host pointers
    T *dVel;
    hipEvent_t  event;
    unsigned int offset;
    unsigned int numBodies;
};


template <typename T>
__device__ typename vec3<T>::Type
bodyBodyInteraction(typename vec3<T>::Type ai,
                    typename vec4<T>::Type bi,
                    typename vec4<T>::Type bj)
{
    typename vec3<T>::Type r;

    // r_ij  [3 FLOPS]
    r.x = bj.x - bi.x;
    r.y = bj.y - bi.y;
    r.z = bj.z - bi.z;

    // distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
    T distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
    distSqr += getSofteningSquared<T>();

    // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
    T invDist = rsqrt_T(distSqr);
    T invDistCube =  invDist * invDist * invDist;

    // s = m_j * invDistCube [1 FLOP]
    T s = bj.w * invDistCube;

    // a_i =  a_i + s * r_ij [6 FLOPS]
    ai.x += r.x * s;
    ai.y += r.y * s;
    ai.z += r.z * s;

    return ai;
}

__device__ half2_3
bodyBodyInteraction_half2( half2_3 ai,
                     half2_4 bi,
                     half2_4 bj)
{
     half2_3 r;

    // r_ij  [3 FLOPS]
    r.x = bj.x - bi.x;
    r.y = bj.y - bi.y;
    r.z = bj.z - bi.z;

    // distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
    half2 distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
    distSqr += __float2half2_rn(getSofteningSquared<float>());

    // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
    half2 invDist = fast_h2rsqrt(distSqr);
    half2 invDistCube =  invDist * invDist * invDist;

    // s = m_j * invDistCube [1 FLOP]
    half2 s = bj.w * invDistCube;

    // a_i =  a_i + s * r_ij [6 FLOPS]
    ai.x += r.x * s;
    ai.y += r.y * s;
    ai.z += r.z * s;

    return ai;
}


template <typename T>
__device__ typename vec3<T>::Type
computeBodyAccel(typename vec4<T>::Type bodyPos,
                 typename vec4<T>::Type *positions,
                 int numTiles, cg::thread_block cta)
{
    typename vec4<T>::Type *sharedPos = SharedMemory<typename vec4<T>::Type>();

    typename vec3<T>::Type acc = {0.0f, 0.0f, 0.0f};

    for (int tile = 0; tile < numTiles; tile++)
    {
        sharedPos[threadIdx.x] = positions[tile * blockDim.x + threadIdx.x];

        cg::sync(cta);

        // This is the "tile_calculation" from the GPUG3 article.
#pragma unroll 128

        for (unsigned int counter = 0; counter < blockDim.x; counter++)
        {
            acc = bodyBodyInteraction<T>(acc, bodyPos, sharedPos[counter]);
        }

        cg::sync(cta);
    }

    return acc;
}


__device__ half2_3
computeBodyAccel_half2(half2_4 bodyPos,
                 typename vec4<float>::Type *positions,
                 int numTiles, cg::thread_block cta)
{
    half2_4 *sharedPos = SharedMemory<half2_4>();

    half2_3 acc;// = {0.0f, 0.0f, 0.0f};
    acc.x = __float2half2_rn(0.0);
    acc.y = __float2half2_rn(0.0);
    acc.z = __float2half2_rn(0.0);
    for (int tile = 0; tile < numTiles; tile++)
    {
        sharedPos[2*threadIdx.x].x = __float2half2_rn(positions[tile * blockDim.x + 2*threadIdx.x].x);
	    	sharedPos[2*threadIdx.x].y = __float2half2_rn(positions[tile * blockDim.x + 2*threadIdx.x].y);
	    	sharedPos[2*threadIdx.x].z = __float2half2_rn(positions[tile * blockDim.x + 2*threadIdx.x].z);
	    	sharedPos[2*threadIdx.x].w = __float2half2_rn(positions[tile * blockDim.x + 2*threadIdx.x].w);

        sharedPos[2*threadIdx.x+1].x = __float2half2_rn(positions[tile * blockDim.x + 2*threadIdx.x+1].x);
	    	sharedPos[2*threadIdx.x+1].y = __float2half2_rn(positions[tile * blockDim.x + 2*threadIdx.x+1].y);
	    	sharedPos[2*threadIdx.x+1].z = __float2half2_rn(positions[tile * blockDim.x + 2*threadIdx.x+1].z);
	    	sharedPos[2*threadIdx.x+1].w = __float2half2_rn(positions[tile * blockDim.x + 2*threadIdx.x+1].w);

        cg::sync(cta);

        // This is the "tile_calculation" from the GPUG3 article.
#pragma unroll 128

        for (unsigned int counter = 0; counter < blockDim.x; counter++)
        {
            acc = bodyBodyInteraction_half2(acc, bodyPos, sharedPos[counter]);
        }

        cg::sync(cta);
    }

    return acc;
}

template<typename T>
__global__ void
integrateBodies(typename vec4<T>::Type *__restrict__ newPos,
                typename vec4<T>::Type *__restrict__ oldPos,
                typename vec4<T>::Type *vel,
                unsigned int deviceOffset, unsigned int deviceNumBodies,
                float deltaTime, float damping, int numTiles)
{

#ifdef SCATTER
  if( blockIdx.x %10 < SPEED){ //speed from 1- 10 .  blockIdx.x %10 < SPEED if need more resolution.
#else
  if( blockIdx.x < SPEED){
#endif
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();

    if (threadIdx.x >= blockDim.x/2)
    {
        return;
    }
    int index = blockIdx.x * blockDim.x + 2*threadIdx.x;
    int index1 = blockIdx.x * blockDim.x + 2*threadIdx.x+1;


//    typename vec4<T>::Type position = oldPos[deviceOffset + index];

	 half2_4 position;
	  position.x = __floats2half2_rn(oldPos[deviceOffset + index].x,oldPos[deviceOffset + index1].x);
	   position.y = __floats2half2_rn(oldPos[deviceOffset + index].y,oldPos[deviceOffset + index1].y);
	    position.z = __floats2half2_rn(oldPos[deviceOffset + index].z,oldPos[deviceOffset + index1].z);
	     position.w = __floats2half2_rn(oldPos[deviceOffset + index].w,oldPos[deviceOffset + index1].w);
  /*
  if(index==0){
      printf("\device \n");
      printf("%f %f %f %f \n", __half2float(position.x.x),__half2float(position.y.x),__half2float(position.z.x),__half2float(position.w.x) );
      printf("%f %f %f %f \n", __half2float(position.x.y),__half2float(position.y.y),__half2float(position.z.y),__half2float(position.w.y) );

    }
    */
   // typename vec3<T>::Type accel = computeBodyAccel<T>(position, oldPos, numTiles, cta);

   half2_3 accel = computeBodyAccel_half2(position,
                                                       oldPos,
                                                       numTiles, cta);


    // acceleration = force / mass;
    // new velocity = old velocity + acceleration * deltaTime
    // note we factor out the body's mass from the equation, here and in bodyBodyInteraction
    // (because they cancel out).  Thus here force == acceleration
    // typename vec4<T>::Type velocity = vel[deviceOffset + index];
	   half2_4 velocity ;
	    velocity.x = __floats2half2_rn(vel[deviceOffset + index].x,vel[deviceOffset + index1].x);
	     velocity.y = __floats2half2_rn(vel[deviceOffset + index].y,vel[deviceOffset + index1].y);
	      velocity.z = __floats2half2_rn(vel[deviceOffset + index].z,vel[deviceOffset + index1].z);
	       velocity.w = __floats2half2_rn(vel[deviceOffset + index].w,vel[deviceOffset + index1].w);
/*
  if(index==0){
      printf("\device velocity\n");
      printf("%f %f %f %f \n", __half2float(velocity.x.x),__half2float(velocity.y.x),__half2float(velocity.z.x),__half2float(velocity.w.x) );
      printf("%f %f %f %f \n", __half2float(velocity.x.y),__half2float(velocity.y.y),__half2float(velocity.z.y),__half2float(velocity.w.y) );

    }
*/
    velocity.x += accel.x * __float2half2_rn(deltaTime);
    velocity.y += accel.y * __float2half2_rn(deltaTime);
    velocity.z += accel.z * __float2half2_rn(deltaTime);

    velocity.x *= __float2half2_rn(damping);
    velocity.y *= __float2half2_rn(damping);
    velocity.z *= __float2half2_rn(damping);

    // new position = old position + velocity * deltaTime
    position.x += velocity.x * __float2half2_rn(deltaTime);
    position.y += velocity.y * __float2half2_rn(deltaTime);
    position.z += velocity.z * __float2half2_rn(deltaTime);

    // store new position and velocity
    //newPos[deviceOffset + index] = position;
    //vel[deviceOffset + index]    = velocity;
    float2 temp = __half22float2(position.x);
    newPos[deviceOffset + index].x = temp.x;
    newPos[deviceOffset + index1].x = temp.y;

 	  temp = __half22float2(position.y);
    newPos[deviceOffset + index].y = temp.x;
    newPos[deviceOffset + index1].y = temp.y;

    temp = __half22float2(position.z);
    newPos[deviceOffset + index].z = temp.x;
    newPos[deviceOffset + index1].z = temp.y;

    temp = __half22float2(position.w);
    newPos[deviceOffset + index].w = temp.x;
    newPos[deviceOffset + index1].w = temp.y;

    temp = __half22float2(velocity.x);
    vel[deviceOffset + index].x = temp.x;
    vel[deviceOffset + index1].x = temp.y;

    temp = __half22float2(velocity.y);
    vel[deviceOffset + index].y = temp.x;
    vel[deviceOffset + index1].y = temp.y;


    temp = __half22float2(velocity.z);
    vel[deviceOffset + index].z = temp.x;
    vel[deviceOffset + index1].z = temp.y;


    temp = __half22float2(velocity.w);
    vel[deviceOffset + index].w = temp.x;
    vel[deviceOffset + index1].w = temp.y;


  } else {
    //if(blockIdx.x %10 == 0 && threadIdx.x ==0)
    //  printf("doing float %d\n", blockIdx.x );
    //doing float comp
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= deviceNumBodies)
    {
        return;
    }

    typename vec4<T>::Type position = oldPos[deviceOffset + index];

    typename vec3<T>::Type accel = computeBodyAccel<T>(position,
                                                       oldPos,
                                                       numTiles, cta);

    // acceleration = force / mass;
    // new velocity = old velocity + acceleration * deltaTime
    // note we factor out the body's mass from the equation, here and in bodyBodyInteraction
    // (because they cancel out).  Thus here force == acceleration
    typename vec4<T>::Type velocity = vel[deviceOffset + index];

    velocity.x += accel.x * deltaTime;
    velocity.y += accel.y * deltaTime;
    velocity.z += accel.z * deltaTime;

    velocity.x *= damping;
    velocity.y *= damping;
    velocity.z *= damping;

    // new position = old position + velocity * deltaTime
    position.x += velocity.x * deltaTime;
    position.y += velocity.y * deltaTime;
    position.z += velocity.z * deltaTime;

    // store new position and velocity
    newPos[deviceOffset + index] = position;
    vel[deviceOffset + index]    = velocity;

  }
}

template <typename T>
void integrateNbodySystem(DeviceData<T> *deviceData,
                          hipGraphicsResource **pgres,
                          unsigned int currentRead,
                          float deltaTime,
                          float damping,
                          unsigned int numBodies,
                          unsigned int numDevices,
                          int blockSize,
                          bool bUsePBO)
{
    if (bUsePBO)
    {
        checkCudaErrors(cudaGraphicsResourceSetMapFlags(pgres[currentRead], cudaGraphicsMapFlagsReadOnly));
        checkCudaErrors(cudaGraphicsResourceSetMapFlags(pgres[1-currentRead], cudaGraphicsMapFlagsWriteDiscard));
        checkCudaErrors(hipGraphicsMapResources(2, pgres, 0));
        size_t bytes;
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&(deviceData[0].dPos[currentRead]), &bytes, pgres[currentRead]));
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&(deviceData[0].dPos[1-currentRead]), &bytes, pgres[1-currentRead]));
    }

    for (unsigned int dev = 0; dev != numDevices; dev++)
    {
        if (numDevices > 1)
        {
            hipSetDevice(dev);
        }

        int numBlocks = (deviceData[dev].numBodies + blockSize-1) / blockSize;
        int numTiles = (numBodies + blockSize - 1) / blockSize;
        int sharedMemSize = blockSize * 4 * sizeof(T); // 4 floats for pos
/*
        integrateBodies<T><<< numBlocks, blockSize, sharedMemSize >>>
        ((typename vec4<T>::Type *)deviceData[dev].dPos[1-currentRead],
         (typename vec4<T>::Type *)deviceData[dev].dPos[currentRead],
         (typename vec4<T>::Type *)deviceData[dev].dVel,
         deviceData[dev].offset, deviceData[dev].numBodies,
         deltaTime, damping, numTiles);
    */
        integrateBodies<float><<< numBlocks, blockSize, sharedMemSize >>>
        ((typename vec4<float>::Type *)deviceData[dev].dPos[1-currentRead],
         (typename vec4<float>::Type *)deviceData[dev].dPos[currentRead],
         (typename vec4<float>::Type *)deviceData[dev].dVel,
         deviceData[dev].offset, deviceData[dev].numBodies,
         deltaTime, damping, numTiles);

        if (numDevices > 1)
        {
            checkCudaErrors(hipEventRecord(deviceData[dev].event));
            // MJH: Hack on older driver versions to force kernel launches to flush!
            hipStreamQuery(0);
        }

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    if (numDevices > 1)
    {
        for (unsigned int dev = 0; dev < numDevices; dev++)
        {
            checkCudaErrors(hipEventSynchronize(deviceData[dev].event));
        }
    }

    if (bUsePBO)
    {
        checkCudaErrors(hipGraphicsUnmapResources(2, pgres, 0));
    }
}


// Explicit specializations needed to generate code
template void integrateNbodySystem<float>(DeviceData<float> *deviceData,
                                          hipGraphicsResource **pgres,
                                          unsigned int currentRead,
                                          float deltaTime,
                                          float damping,
                                          unsigned int numBodies,
                                          unsigned int numDevices,
                                          int blockSize,
                                          bool bUsePBO);

template void integrateNbodySystem<double>(DeviceData<double> *deviceData,
                                           hipGraphicsResource **pgres,
                                           unsigned int currentRead,
                                           float deltaTime,
                                           float damping,
                                           unsigned int numBodies,
                                           unsigned int numDevices,
                                           int blockSize,
                                           bool bUsePBO);
