#include "hip/hip_runtime.h"
#include "stdlib.h"
#include <fstream>
#include <iostream>
#include <cstddef>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define MAX_LOOP 1000
#define MAX_DIFF 0.15f
#define NUM_JOINTS 3
#define PI 3.14159265358979f
#define NUM_JOINTS_P1 (NUM_JOINTS + 1)

using namespace std;

#ifdef SLOW_MATH
#include "../include/cuda_math.cuh"
#else
#include "../include/fast_math.cuh"
#endif

#define SCATTER

using namespace std;
__global__ void invkin_kernel(float *xTarget_in, float *yTarget_in, float *angles, int size, float err_thresh, int speed) {
#ifdef SCATTER
	if(blockIdx.x %100 < speed) {
#else
	if(blockIdx.x < speed) {
#endif

	if(threadIdx.x<blockDim.x/2){
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + 2*threadIdx.x;
	if(idx < size)
	{
    	half2 angle_out[NUM_JOINTS];

			half zero = 0.0f;
			half one = 1.f;
			half minus_one = -1.f;

    	for(int i = 0; i < NUM_JOINTS; i++)
    	{
  			angle_out[i] = __float2half2_rn(0.0);
				//angle_out[i].x = 0.f;
				//angle_out[i].y = 0.f;
    	}

    	half max_err 	= err_thresh * (float)(NUM_JOINTS);
    	half err 		= max_err + one; // initialize error to something greater than error threshold


		// Initialize x and y data
		half2 xData[NUM_JOINTS_P1];
		half2 yData[NUM_JOINTS_P1];

		for (int i = 0 ; i < NUM_JOINTS_P1; i++)
		{
			xData[i] = __float2half2_rn((float)i);
			yData[i] = __float2half2_rn(0.f);
		}

		half2 xTarget_in_temp = __floats2half2_rn(xTarget_in[idx],xTarget_in[idx+1]);
		half2 yTarget_in_temp = __floats2half2_rn(yTarget_in[idx],yTarget_in[idx+1]);


		//half minus_one = -1.0f;

		half2 pe_x = xData[NUM_JOINTS];
		half2 pe_y = yData[NUM_JOINTS];

		for(int curr_loop = 0; curr_loop < MAX_LOOP; curr_loop++)
		{
			for (int iter = NUM_JOINTS; iter > 0; iter--)
			{
				half2 pc_x = xData[iter-1];
				half2 pc_y = yData[iter-1];
				half2 diff_pe_pc_x = pe_x - pc_x;
				half2 diff_pe_pc_y = pe_y - pc_y;
//				half2 diff_tgt_pc_x = xTarget_in[idx] - pc_x;
//				half2 diff_tgt_pc_y = yTarget_in[idx] - pc_y;
				half2 diff_tgt_pc_x = xTarget_in_temp - pc_x;
				half2 diff_tgt_pc_y = yTarget_in_temp - pc_y;
				half2 len_diff_pe_pc = fast_h2sqrt (diff_pe_pc_x * diff_pe_pc_x + diff_pe_pc_y * diff_pe_pc_y);
				half2 len_diff_tgt_pc = fast_h2sqrt (diff_tgt_pc_x * diff_tgt_pc_x + diff_tgt_pc_y * diff_tgt_pc_y);
				half2 a_x = diff_pe_pc_x * fast_h2rcp(len_diff_pe_pc);
				half2 a_y = diff_pe_pc_y * fast_h2rcp(len_diff_pe_pc);
				half2 b_x = diff_tgt_pc_x * fast_h2rcp(len_diff_tgt_pc);
				half2 b_y = diff_tgt_pc_y * fast_h2rcp(len_diff_tgt_pc);
				half2 a_dot_b = a_x * b_x + a_y * b_y;


				//float2 a_dot_b_float  = __half22float2(a_dot_b);

				if (a_dot_b.x > one) {
					a_dot_b.x = one ;
				}
				if (a_dot_b.x < minus_one) {
					a_dot_b.x = minus_one ;
				}
				if (a_dot_b.y > one) {
					a_dot_b.y = one ;
				}
				if (a_dot_b.y < minus_one) {
					a_dot_b.y = minus_one ;
				}

		/*
				if (a_dot_b > 1.f)
					a_dot_b = 1.f;
				else if (a_dot_b < -1.f)
					a_dot_b = -1.f;
*/


				//float2 a_dot_b_float = __half22float2(a_dot_b);
				//half2 angle =__floats2half2_rn (acosf(a_dot_b_float.x) * (180.f / PI), acosf(a_dot_b_float.x) * (180.f / PI));
				//angle.x = acosf(a_dot_b_float.x) * (180.f / PI);
				//angle.y = acosf(a_dot_b_float.y) * (180.f / PI);
				half2 angle = fast_h2acos(a_dot_b) * 57.29578;//(180.f / PI);
				// Determine angle direction

				half2 direction = a_x * b_y - a_y * b_x;
				if (direction.x < zero)

					angle.x = -angle.x ;
				if (direction.y < zero)
					angle.y = -angle.y;

				// Make the result look more natural (these checks may be omitted)
				// if (angle > 30.f)
				// 	angle = 30.f;
				// else if (angle < -30.f)
				// 	angle = -30.f;
				// Save angle
				angle_out[iter - 1] = angle;
				for (int i = 0; i < NUM_JOINTS; i++)
				{
					if(i < NUM_JOINTS - 1)
					{
						angle_out[i+1] += angle_out[i];
						//angle_out[i+1].y += angle_out[i].y;
					}
				}
			}// loop NUM_JOINTS
		}// loop 1k


		float2 angle_0 = __half22float2(angle_out[0]);
		float2 angle_1 = __half22float2(angle_out[1]);
		float2 angle_2 = __half22float2(angle_out[2]);

		angles[idx * NUM_JOINTS + 0] = angle_0.x;
		angles[idx * NUM_JOINTS + 1] = angle_1.x;
		angles[idx * NUM_JOINTS + 2] = angle_2.x;

		angles[(idx+1) * NUM_JOINTS + 0] = angle_0.y;
		angles[(idx+1) * NUM_JOINTS + 1] = angle_1.y;
		angles[(idx+1) * NUM_JOINTS + 2] = angle_2.y;
	}


} //end if(threadIdx.x<512/2)
else return;
} else {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	if(idx < size)
	{
//		float parrotInput[2];
//    	float parrotOutput[3];

    	float angle_out[NUM_JOINTS];

    	for(int i = 0; i < NUM_JOINTS; i++)
    	{
  			angle_out[i] = 0.0;
    	}

    	float max_err 	= err_thresh * (float)(NUM_JOINTS);
    	float err 		= max_err + 1.f; // initialize error to something greater than error threshold

/* parrot not used
    	parrotInput[0] = xTarget_in[idx];
    	parrotInput[1] = yTarget_in[idx];

#pragma parrot(input, "invkin_kernel", [2]<-1.0; 1.0>parrotInput)
*/
		//float max_err = err_thresh * (float)(NUM_JOINTS);
		//float err = max_err + 1.f;

		// Initialize x and y data
		float xData[NUM_JOINTS_P1];
		float yData[NUM_JOINTS_P1];

		for (int i = 0 ; i < NUM_JOINTS_P1; i++)
		{
			xData[i] = i;
			yData[i] = 0.f;
		}

		for(int curr_loop = 0; curr_loop < MAX_LOOP; curr_loop++)
		{
			for (int iter = NUM_JOINTS; iter > 0; iter--)
			{
				float pe_x = xData[NUM_JOINTS];
				float pe_y = yData[NUM_JOINTS];
				float pc_x = xData[iter-1];
				float pc_y = yData[iter-1];
				float diff_pe_pc_x = pe_x - pc_x;
				float diff_pe_pc_y = pe_y - pc_y;
				float diff_tgt_pc_x = xTarget_in[idx] - pc_x;
				float diff_tgt_pc_y = yTarget_in[idx] - pc_y;
				float len_diff_pe_pc = sqrt(diff_pe_pc_x * diff_pe_pc_x + diff_pe_pc_y * diff_pe_pc_y);
				float len_diff_tgt_pc = sqrt(diff_tgt_pc_x * diff_tgt_pc_x + diff_tgt_pc_y * diff_tgt_pc_y);
				float a_x = diff_pe_pc_x / len_diff_pe_pc;
				float a_y = diff_pe_pc_y / len_diff_pe_pc;
				float b_x = diff_tgt_pc_x / len_diff_tgt_pc;
				float b_y = diff_tgt_pc_y / len_diff_tgt_pc;
				float a_dot_b = a_x * b_x + a_y * b_y;
				if (a_dot_b > 1.f)
					a_dot_b = 1.f;
				else if (a_dot_b < -1.f)
					a_dot_b = -1.f;
				float angle = acos(a_dot_b) * (180.f / PI);
				// Determine angle direction
				float direction = a_x * b_y - a_y * b_x;
				if (direction < 0.f)
					angle = -angle;
				// Make the result look more natural (these checks may be omitted)
				// if (angle > 30.f)
				// 	angle = 30.f;
				// else if (angle < -30.f)
				// 	angle = -30.f;
				// Save angle
				angle_out[iter - 1] = angle;
				for (int i = 0; i < NUM_JOINTS; i++)
				{
					if(i < NUM_JOINTS - 1)
					{
						angle_out[i+1] += angle_out[i];
					}
				}
			}
		}

/*		parrot : not used
		parrotOutput[0] = angle_out[0] / 30.0;
		parrotOutput[1] = angle_out[1] / 30.0;
		parrotOutput[2] = angle_out[2] / 30.0;

#pragma parrot(output, "invkin_kernel", [3]<-1.0; 1.0>parrotOutput)

		angle_out[0] = parrotOutput[0] * 30.0;
		angle_out[1] = parrotOutput[1] * 30.0;
		angle_out[2] = parrotOutput[2] * 30.0;
*/
		angles[idx * NUM_JOINTS + 0] = angle_out[0];
		angles[idx * NUM_JOINTS + 1] = angle_out[1];
		angles[idx * NUM_JOINTS + 2] = angle_out[2];
	}
}
}

int main(int argc, char* argv[])
{
	int speed = 50;
	std::cout << "# Speed = " << speed << std::endl;

	if(argc != 4)
	{
		std::cerr << "Usage: ./invkin.out <input file coefficients> <output file> <error threshold>" << std::endl;
		exit(EXIT_FAILURE);
	}

	float* xTarget_in_h;
	float* yTarget_in_h;
	float* angle_out_h;

	hipError_t cudaStatus;

	int data_size = 0;

	// process the files
	ifstream coordinate_in_file (argv[1]);
	ofstream angle_out_file (argv[2]);
	float err_thresh = atof(argv[3]);


	if(coordinate_in_file.is_open())
	{
		coordinate_in_file >> data_size;
		std::cout << "# Data Size = " << data_size << std::endl;
	}

	// allocate the memory
	xTarget_in_h = new (nothrow) float[data_size];
	if(xTarget_in_h == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}
	yTarget_in_h = new (nothrow) float[data_size];
	if(yTarget_in_h == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}
	angle_out_h = new (nothrow) float[data_size*NUM_JOINTS];
	if(angle_out_h == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}


	// Prepare
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	// add data to the arrays
	float xTarget_tmp, yTarget_tmp;
	int coeff_index = 0;
	while(coeff_index < data_size)
	{
		coordinate_in_file >> xTarget_tmp >> yTarget_tmp;

		for(int i = 0; i < NUM_JOINTS ; i++)
		{
			angle_out_h[coeff_index * NUM_JOINTS + i] = 0.0;
		}

		xTarget_in_h[coeff_index] = xTarget_tmp;
		yTarget_in_h[coeff_index++] = yTarget_tmp;
	}


	std::cout << "# Coordinates are read from file..." << std::endl;

	// memory allocations on the host
	float 	*xTarget_in_d,
			*yTarget_in_d;
	float 	*angle_out_d;

	hipMalloc((void**) &xTarget_in_d, data_size * sizeof(float));
	hipMalloc((void**) &yTarget_in_d, data_size * sizeof(float));
	hipMalloc((void**) &angle_out_d,  data_size * NUM_JOINTS * sizeof(float));

	std::cout << "# Memory allocation on GPU is done..." << std::endl;

	hipMemcpy(xTarget_in_d, xTarget_in_h, data_size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(yTarget_in_d, yTarget_in_h, data_size * sizeof(float), hipMemcpyHostToDevice);

	std::cout << "# Data are transfered to GPU..." << std::endl;

	dim3 dimBlock	( 512, 1 );
	dim3 dimGrid	( data_size / 512, 1 );


	hipEventRecord(start, 0);

#pragma parrot.start("invkin_kernel")

	invkin_kernel<<<dimGrid, dimBlock>>>(xTarget_in_d, yTarget_in_d, angle_out_d, data_size, err_thresh, speed);

#pragma parrot.end("invkin_kernel")

	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
       	std::cout << "Something was wrong! Error code: " << cudaStatus << std::endl;
    }

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	std::cout << "# Elapsed Time in `nrpoly3` kernel = " << elapsedTime << std::endl;
	std::cout << "# GPU computation is done ..." << std::endl;

	hipMemcpy(angle_out_h, angle_out_d, data_size * NUM_JOINTS * sizeof(float), hipMemcpyDeviceToHost);

	for(int i = 0; i < data_size; i++)
	{
		// angle_out_file << xTarget_in_h[i] << " " << yTarget_in_h[i] << " ";
		//compare output, not need to store this
		for(int j = 0 ; j < NUM_JOINTS; j++)
		{
			angle_out_file << angle_out_h[i * NUM_JOINTS + j] << " ";
		}
		angle_out_file << std::endl;
	}

	// close files
	coordinate_in_file.close();
	angle_out_file.close();

	// de-allocate the memory
	delete[] xTarget_in_h;
	delete[] yTarget_in_h;
	delete[] angle_out_h;

	// de-allocate cuda memory
	hipFree(xTarget_in_d);
	hipFree(yTarget_in_d);
	hipFree(angle_out_d);

	std::cout << "Thank you..." << std::endl;
}