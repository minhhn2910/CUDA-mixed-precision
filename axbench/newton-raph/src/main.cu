// Designed by: Amir Yazdanbakhsh
// Date: March 26th - 2015
// Alternative Computing Technologies Lab.
// Georgia Institute of Technology


#include "stdlib.h"
#include <iostream>
#include <fstream>
#include <cstddef>

// Cuda Libraries
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

//#define MAX_LOOP 5
#define MAX_LOOP 20 //more loops, even number

#define MAX_DIFF 0.15f

#define F(x) 	(A_coeff[idx]*x*x*x)+(B_coeff[idx]*x*x)+(C_coeff[idx]*x)+D_coeff[idx]
#define FD(x) 	(3.0*A_coeff[idx]*x*x)+(2.0*B_coeff[idx]*x)+(C_coeff[idx])

using namespace std;

__global__ void nrpol3_kernel(float *A_coeff, float *B_coeff, float *C_coeff, float *D_coeff, float *x0_in, float *root, int size, float err_thresh)
{

	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	if(idx < size)
	{

/*		float parrotInput[5];
    	float parrotOutput[1];

    	parrotInput[0] = A_coeff[idx] / 20.0;
    	parrotInput[1] = B_coeff[idx] / 20.0;
    	parrotInput[2] = C_coeff[idx] / 20.0;
    	parrotInput[3] = D_coeff[idx] / 4000.0;
    	parrotInput[4] = x0_in[idx]   / 15000.0;
			parrot is not used, test on real device;

#pragma parrot(input, "nrpol3_kernel", [5]parrotInput)
*/
		float curr_err = MAX_DIFF * err_thresh;
		float x0 = x0_in[idx];
		float x1, fx, fdx;
		float temp_result = 0;//store temp_res in loop
		for (int i = 0; i < MAX_LOOP; i++) {
			//fx  = F(x0);
			fx = (A_coeff[idx]*x0*x0*x0)+(B_coeff[idx]*x0*x0)+(C_coeff[idx]*x0)+D_coeff[idx];
			//fdx = FD(x0);
			fdx = (3.0*A_coeff[idx]*x0*x0)+(2.0*B_coeff[idx]*x0)+(C_coeff[idx]);
			x1  = x0 - (fx/fdx);
			if (fabs((x1-x0) / x1) < curr_err) {
				curr_err = fabs((x1-x0) / x1);
				temp_result = x1;
			}
			x0 = x1;
		}
//		parrotOutput[0] = root[idx] / 20.0;
//#pragma parrot(output, "nrpol3_kernel", [1]<-2.0; 2.0>parrotOutput)
		root[idx] = temp_result;//parrotOutput[0] * 20.0;

	}
}

int main(int argc, char* argv[])
{
	if(argc != 4)
	{
		std::cerr << "Usage: ./nrpoly3.out <input file coefficients> <output file> <error threshold>" << std::endl;
		exit(EXIT_FAILURE);
	}

	float* A_coeff;
	float* B_coeff;
	float* C_coeff;
	float* D_coeff;
	float* x0;
	float* root;

	hipError_t cudaStatus;

	int data_size = 0;

	// process the files
	ifstream coeff_in_file (argv[1]);
	ofstream root_out_file (argv[2]);
	float err_thresh = atof(argv[3]);


	if(coeff_in_file.is_open())
	{
		coeff_in_file >> data_size;
		std::cout << "# Data Size = " << data_size << std::endl;
	}

	// allocate the memory
	A_coeff = new (nothrow) float[data_size];
	if(A_coeff == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}
	B_coeff = new (nothrow) float[data_size];
	if(B_coeff == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}
	C_coeff = new (nothrow) float[data_size];
	if(C_coeff == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}
	D_coeff = new (nothrow) float[data_size];
	if(D_coeff == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}
	x0 = new (nothrow) float[data_size];
	if(x0 == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}
	root = new (nothrow) float[data_size];
	if(root == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}


	// Prepare
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	// add data to the arrays
	float A_tmp, B_tmp, C_tmp, D_tmp, x0_tmp;
	int coeff_index = 0;
	while(coeff_index < data_size)
	{
		coeff_in_file >> A_tmp >> B_tmp >> C_tmp >> D_tmp >> x0_tmp;

		root 	[coeff_index]   = 0;
		A_coeff	[coeff_index] 	= A_tmp;
		B_coeff	[coeff_index] 	= B_tmp;
		C_coeff	[coeff_index]	= C_tmp;
		D_coeff	[coeff_index] 	= D_tmp;
		x0 		[coeff_index++] = x0_tmp;
	}


	std::cout << "# Coefficients are read from file..." << std::endl;

	// memory allocations on the host
	float 	*A_coeff_d,
			*B_coeff_d,
			*C_coeff_d,
			*D_coeff_d,
			*x0_d;
	float 	* root_d;

	hipMalloc((void**) &A_coeff_d, data_size * sizeof(float));
	hipMalloc((void**) &B_coeff_d, data_size * sizeof(float));
	hipMalloc((void**) &C_coeff_d, data_size * sizeof(float));
	hipMalloc((void**) &D_coeff_d, data_size * sizeof(float));
	hipMalloc((void**) &x0_d, 		data_size * sizeof(float));
	hipMalloc((void**) &root_d,	data_size * sizeof(float));

	std::cout << "# Memory allocation on GPU is done..." << std::endl;

	hipMemcpy(A_coeff_d, A_coeff, data_size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_coeff_d, B_coeff, data_size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(C_coeff_d, C_coeff, data_size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(D_coeff_d, D_coeff, data_size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(x0_d, 	  x0,    data_size * sizeof(float), hipMemcpyHostToDevice);

	std::cout << "# Data are transfered to GPU..." << std::endl;

	dim3 dimBlock	( 512, 1 );
	dim3 dimGrid	( data_size / 512, 1 );


	hipEventRecord(start, 0);

#pragma parrot.start("nrpol3_kernel")

	nrpol3_kernel<<<dimGrid, dimBlock>>>(A_coeff_d, B_coeff_d, C_coeff_d, D_coeff_d, x0_d, root_d, data_size, err_thresh);

#pragma parrot.end("nrpol3_kernel")

	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
       	std::cout << "Something was wrong! Error code: " << cudaStatus << std::endl;
    }

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	std::cout << "# Elapsed Time in `nrpoly3` kernel = " << elapsedTime << std::endl;
	std::cout << "# GPU computation is done ..." << std::endl;

	hipMemcpy( root, root_d, data_size * sizeof(float), hipMemcpyDeviceToHost);

	for(int i = 0; i < data_size; i++)
	{
		root_out_file << root[i] << std::endl;
	}

	// close files
	root_out_file.close();
	coeff_in_file.close();

	// de-allocate the memory
	delete[] A_coeff;
	delete[] B_coeff;
	delete[] C_coeff;
	delete[] D_coeff;
	delete[] x0;
	delete[] root;

	// de-allocate cuda memory
	hipFree(A_coeff_d);
	hipFree(B_coeff_d);
	hipFree(C_coeff_d);
	hipFree(D_coeff_d);
	hipFree(x0_d);
	hipFree(root_d);

	std::cout << "Thank you..." << std::endl;
}
