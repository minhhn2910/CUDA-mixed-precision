
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#define TYPE1 double
#define TYPE2 float
#define TYPE3 float
#define TYPE4 double
#define SPEED 0
double fun_ref( double x){
  int k, n = 5;
  double t1;
  double d1 = 1.0;
  t1 = x;
  for ( k = 1; k <= n; k++ )
    {
      d1 = 2.0 * d1;
      t1 = t1+ sin(d1 * x)/d1;
    }
    return t1;
}
__global__ void fun_gpu(double x[], double y[], int nthreads, int speed){
  //y = fun(x)
  //speed = %
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if(blockIdx.x % 10< speed){
    int k, n = 5;
    if (tid < nthreads) {
      double t1;
      float d1 = 1.0;
      float x_temp = x[tid];
      t1 = x[tid];
      for ( k = 1; k <= n; k++ )
        {
          d1 = 2.0 * d1;
          double sin_res = sin(d1 * x_temp);
          t1 = t1 + sin_res/d1;
        }
      y[tid] = t1;
  }

  }
  else{

        int k, n = 5;
        if (tid < nthreads) {
          double t1;
          double d1 = 1.0;
          double x_temp = x[tid];
          t1 = x[tid];
          for ( k = 1; k <= n; k++ )
            {
              d1 = 2.0 * d1;
              double sin_res = sin(d1 * x_temp);
              t1 = t1 + sin_res/d1;
            }
          y[tid] = t1;
        }
 }

}


int main( int argc, char **argv) {
  int i,n = 1000000;
  double h, t1, t2, dppi;
  double s1;
  //cuda def
  hipEvent_t start, stop;
  float elapsedTime;
  int speed = atoi(argv[1]);
  printf("running with speed %d \n", speed);
  double *d_x, *d_y, *h_x, *h_y ;
  size_t size = n*sizeof(double);

  h_x = (double*) malloc(size);
  h_y = (double*) malloc(size);
  hipMalloc(&d_x, size);
  hipMalloc(&d_y, size);


  t1 = -1.0;
  dppi = acos(t1);
  s1 = 0.0;
  t1 = 0.0;
  h = dppi / n;
  for ( i = 1; i <= n; i++){
    h_x[i-1] = i * h;
  }
    /* Copy vectors from host memory to device memory */
  hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

  int threads_per_block = 256;

  int block_count = (n + threads_per_block - 1)/threads_per_block;
  hipEventCreate(&start);
  hipEventRecord(start,0);
  for (int i =0;i < 10; i ++)
    fun_gpu<<<block_count, threads_per_block>>>(d_x, d_y, n,speed);

  hipDeviceSynchronize();
  hipEventCreate(&stop);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start,stop);
  printf("Elapsed time : %f ms\n" ,elapsedTime);
  hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);
  for ( i = 1; i <= n; i++)
    {
      t2 = h_y[i-1];
      s1 = s1 + sqrt(h*h + (t2 - t1) * (t2 - t1));
      t1 = t2;
    }
  double ref_value = 5.7957763224;
  printf("%.10f\n",s1);
  printf("abs err %.8f  rel err %.8f\n", fabs(s1-ref_value), fabs((s1-ref_value)/ref_value) );
  return 0;
}
