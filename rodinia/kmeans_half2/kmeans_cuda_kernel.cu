#include "hip/hip_runtime.h"
#ifndef _KMEANS_CUDA_KERNEL_H_
#define _KMEANS_CUDA_KERNEL_H_

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "kmeans.h"

// FIXME: Make this a runtime selectable variable!
#define ASSUMED_NR_CLUSTERS 32

#define SDATA( index)      CUT_BANK_CHECKER(sdata, index)

// t_features has the layout dim0[points 0-m-1]dim1[ points 0-m-1]...
texture<int, 1, hipReadModeElementType> t_features;
// t_features_flipped has the layout point0[dim 0-n-1]point1[dim 0-n-1]
texture<int, 1, hipReadModeElementType> t_features_flipped;
texture<int, 1, hipReadModeElementType> t_clusters;


__constant__ half c_clusters[ASSUMED_NR_CLUSTERS*34];		/* constant memory for cluster centers */

/* ----------------- invert_mapping() --------------------- */
/* inverts data array from row-major to column-major.

   [p0,dim0][p0,dim1][p0,dim2] ...
   [p1,dim0][p1,dim1][p1,dim2] ...
   [p2,dim0][p2,dim1][p2,dim2] ...
										to
   [dim0,p0][dim0,p1][dim0,p2] ...
   [dim1,p0][dim1,p1][dim1,p2] ...
   [dim2,p0][dim2,p1][dim2,p2] ...
*/
__global__ void invert_mapping(half *input,			/* original */
							   half *output,			/* inverted */
							   int npoints,				/* npoints */
							   int nfeatures)			/* nfeatures */
{
	int point_id = threadIdx.x + blockDim.x*blockIdx.x;	/* id of thread */
	int i;

	if(point_id < npoints){
		for(i=0;i<nfeatures;i++)
			output[point_id + npoints*i] = input[point_id*nfeatures + i];
	}
	return;
}
/* ----------------- invert_mapping() end --------------------- */

/* to turn on the GPU delta and center reduction */
//#define GPU_DELTA_REDUCTION
//#define GPU_NEW_CENTER_REDUCTION


/* ----------------- kmeansPoint() --------------------- */
/* find the index of nearest cluster centers and change membership*/
__global__ void
kmeansPoint(half2  *features,			/* in: [npoints*nfeatures] */
            int     nfeatures,
            int     npoints,
            int     nclusters,
            int    *membership,
			half2  *clusters,
			half2  *block_clusters,
			int    *block_deltas)
{

	// block ID
	const unsigned int block_id = gridDim.x*blockIdx.y+blockIdx.x;
	// point/thread ID
	const unsigned int point_id = block_id*blockDim.x*blockDim.y + threadIdx.x;

	int  index = -1;
	int index2 = -1;
	if (point_id < npoints/2)
	{
		int i, j;
		half2 min_dist = __float2half2_rn(FLT_MAX);
		half2 dist;													/* distance square between a point to cluster center */

		/* find the cluster center id with min distance to pt */
		for (i=0; i<nclusters; i++) {
			int cluster_base_index = i*nfeatures;					/* base index of cluster centers for inverted array */
			half2 ans=__float2half2_rn(0.0);												/* Euclidean distance sqaure */

			for (j=0; j < nfeatures; j++)
			{
				int addr = point_id + j*npoints;					/* appropriate index of data point */
				int temp = tex1Dfetch(t_features,addr);
				half2 diff = ( *(__half2*)(&temp)- __half2half2(c_clusters[cluster_base_index + j]));	/* distance between a data point to cluster centers */
				ans += diff*diff;									/* sum of squares */
			}
			dist = ans;

			/* see if distance is smaller than previous ones:
			if so, change minimum distance and save index of cluster center */
			if (((half*)(&dist))[0] < ((half*)(&min_dist))[0]) {
				((half*)(&min_dist))[0] = ((half*)(&dist))[0] ;
				index    = i;
			}
			if (((half*)(&dist))[1] < ((half*)(&min_dist))[1]) {
				((half*)(&min_dist))[1] = ((half*)(&dist))[1] ;
				index2    = i;
			}
	}


		/* assign the membership to object point_id */
		membership[2*point_id] = index;
		membership[2*point_id+1] = index2;
	}

}
#endif // #ifndef _KMEANS_CUDA_KERNEL_H_
