#include "hip/hip_runtime.h"
#ifndef _KMEANS_CUDA_KERNEL_H_
#define _KMEANS_CUDA_KERNEL_H_

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "kmeans.h"

#define SCATTER

// FIXME: Make this a runtime selectable variable!
#define ASSUMED_NR_CLUSTERS 32

#define SDATA( index)      CUT_BANK_CHECKER(sdata, index)

// t_features has the layout dim0[points 0-m-1]dim1[ points 0-m-1]...
texture<float, 1, hipReadModeElementType> t_features;
// t_features_flipped has the layout point0[dim 0-n-1]point1[dim 0-n-1]
texture<float, 1, hipReadModeElementType> t_features_flipped;
texture<float, 1, hipReadModeElementType> t_clusters;


__constant__ float c_clusters[ASSUMED_NR_CLUSTERS*34];		/* constant memory for cluster centers */

/* ----------------- invert_mapping() --------------------- */
/* inverts data array from row-major to column-major.

   [p0,dim0][p0,dim1][p0,dim2] ...
   [p1,dim0][p1,dim1][p1,dim2] ...
   [p2,dim0][p2,dim1][p2,dim2] ...
										to
   [dim0,p0][dim0,p1][dim0,p2] ...
   [dim1,p0][dim1,p1][dim1,p2] ...
   [dim2,p0][dim2,p1][dim2,p2] ...
*/
__global__ void invert_mapping(float *input,			/* original */
							   float *output,			/* inverted */
							   int npoints,				/* npoints */
							   int nfeatures)			/* nfeatures */
{
	int point_id = threadIdx.x + blockDim.x*blockIdx.x;	/* id of thread */
	int i;

	if(point_id < npoints){
		for(i=0;i<nfeatures;i++)
			output[point_id + npoints*i] = input[point_id*nfeatures + i];
	}
	return;
}
/* ----------------- invert_mapping() end --------------------- */

/* to turn on the GPU delta and center reduction */
//#define GPU_DELTA_REDUCTION
//#define GPU_NEW_CENTER_REDUCTION


/* ----------------- kmeansPoint() --------------------- */
/* find the index of nearest cluster centers and change membership*/
__global__ void
kmeansPoint(float  *features,			/* in: [npoints*nfeatures] */
            int     nfeatures,
            int     npoints,
            int     nclusters,
            int    *membership,
			float  *clusters,
			float  *block_clusters,
			int    *block_deltas,
			int speed
			//,float* distance_vec
			)
{

	// block ID
	const unsigned int block_id = gridDim.x*blockIdx.y+blockIdx.x;
#ifdef SCATTER
  if(block_id %10 < speed){
#else
  if(block_id < speed){
#endif
//if (block_id < speed)
//{
	// point/thread ID
	const unsigned int point_id = block_id*blockDim.x*blockDim.y + threadIdx.x;

	int  index = -1;

	if (point_id < npoints)
	{
		int i, j;
		//float min_dist = FLT_MAX;
		//half2 dist;													/* distance square between a point to cluster center */
		half min_dist = __float2half_rn(FLT_MAX);

		/* find the cluster center id with min distance to pt */
		for (i=0; i<nclusters/2; i++) {
			int cluster_base_index = 2*i*nfeatures;					/* base index of cluster centers for inverted array */
			int cluster_base_index_next = (2*i+1)*nfeatures;					/* base index of cluster centers for inverted array */

			half2 ans=__float2half2_rn(0.0);												/* Euclidean distance sqaure */

			for (j=0; j < nfeatures; j++)
			{
				int addr = point_id + j*npoints;					/* appropriate index of data point */
				//float diff = (tex1Dfetch(t_features,addr) -
				//			  c_clusters[cluster_base_index + j]);	/* distance between a data point to cluster centers */
				half2 c_c = __floats2half2_rn(c_clusters[cluster_base_index + j], c_clusters[cluster_base_index_next + j]);
				half2 feature_temp = __float2half2_rn(tex1Dfetch(t_features,addr));
				half2 diff = (feature_temp-c_c) ;
				//ans += (feature_temp-c_c) * (feature_temp-c_c);
				ans += diff*diff;									/* sum of squares */
			}

			//dist = ans;
			//~ distance_vec[point_id*nclusters + nclusters] = ans;
			/* see if distance is smaller than previous ones:
			if so, change minimum distance and save index of cluster center */
			half high_ = __high2half(ans);
			half low_ = __low2half(ans);
			if (high_ < min_dist) {
				min_dist = high_;
				index    = 2*i;
			}
			if (low_ < min_dist){
				min_dist = low_;
				index    = 2*i+1;
			}

		}

		 membership[point_id] = index;
	}

}else{
	//doing float computation
	const unsigned int point_id = block_id*blockDim.x*blockDim.y + threadIdx.x;

	int  index = -1;

	if (point_id < npoints)
	{
		int i, j;
		float min_dist = FLT_MAX;
		float dist;													/* distance square between a point to cluster center */

		/* find the cluster center id with min distance to pt */
		for (i=0; i<nclusters; i++) {
			int cluster_base_index = i*nfeatures;					/* base index of cluster centers for inverted array */
			float ans=0.0;												/* Euclidean distance sqaure */

			for (j=0; j < nfeatures; j++)
			{
				int addr = point_id + j*npoints;					/* appropriate index of data point */
				float diff = (tex1Dfetch(t_features,addr) -
							  c_clusters[cluster_base_index + j]);	/* distance between a data point to cluster centers */
				ans += diff*diff;									/* sum of squares */
			}
			dist = ans;
			//~ distance_vec[point_id*nclusters + nclusters] = ans;
			/* see if distance is smaller than previous ones:
			if so, change minimum distance and save index of cluster center */
			if (dist < min_dist) {
				min_dist = dist;
				index    = i;
			}

		}

		 membership[point_id] = index;
}

}

}
#endif // #ifndef _KMEANS_CUDA_KERNEL_H_
